#include "hip/hip_runtime.h"
/*
 * All changes to the original code are Copyright DataStax, Inc.
 *
 * Please see the included license file for details.
 */

/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <random>
#include <chrono>
#include <iostream>
#include <vector>
#include <mutex>

#include <cstdint>
#include <cstdlib>

#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_resources.hpp>
#include <raft/core/device_resources_manager.hpp>
#include <raft/random/make_blobs.cuh>

#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <cuvs/neighbors/cagra.hpp>

struct jv_cagra_builder_t {
    raft::device_resources dev_resources;
    std::vector<std::vector<float>> host_vectors;
    int64_t dim;

    jv_cagra_builder_t(int32_t n_nodes, int64_t dim)
        : dev_resources(), dim(dim) {
        host_vectors.reserve(n_nodes);
    }
};

struct jv_cagra_index_t {
    cuvs::neighbors::cagra::index<float, uint32_t> index;

    jv_cagra_index_t(cuvs::neighbors::cagra::index<float, uint32_t>&& index)
        : index(std::move(index)) {}
};

extern "C" {
    jv_cagra_builder_t* create_cagra_builder(int32_t n_nodes, int64_t dim) {
        return new jv_cagra_builder_t(n_nodes, dim);
    }

    void add_node(jv_cagra_builder_t* builder, float* vector) {
        if (builder == nullptr || vector == nullptr) {
            return;
        }
        std::vector<float> vec(builder->dim);
        std::copy(vector, vector + builder->dim, vec.begin());
        builder->host_vectors.push_back(std::move(vec));
    }

    jv_cagra_index_t* build_cagra_index(jv_cagra_builder_t* builder) {
        if (builder == nullptr) {
            return nullptr;
        }

        using namespace cuvs::neighbors;
        int64_t n_nodes = builder->host_vectors.size();
        int64_t dim = builder->dim;

        // Create device matrices
        auto device_vectors = raft::make_device_matrix<float, int64_t>(builder->dev_resources, n_nodes, dim);

        // Copy data to device
        std::vector<float> flattened_vectors;
        for (const auto& vec : builder->host_vectors) {
            flattened_vectors.insert(flattened_vectors.end(), vec.begin(), vec.end());
        }
        raft::copy(device_vectors.data_handle(), flattened_vectors.data(), flattened_vectors.size(), raft::resource::get_cuda_stream(builder->dev_resources));
        builder->dev_resources.sync_stream();

        // Build the index
        // DEMOFIXME: tune index_params instead of using defaults
        cagra::index_params index_params;
        auto index = cagra::build(builder->dev_resources, index_params, raft::make_const_mdspan(device_vectors.view()));
        builder->dev_resources.sync_stream();
        return new jv_cagra_index_t(std::move(index));

        // DEMOFIXME: clean up builder resources
    }

    int32_t* search_cagra_index(jv_cagra_index_t* index, float* query, int32_t topk) {
        if (index == nullptr || query == nullptr) {
            return nullptr;
        }

        raft::common::nvtx::range fun_scope("search_cagra_index(k = %u, dim = %zu)", topk, index->index.dim());

        // Neighbors are returned to host, so we keep them in the page-locked cuda-registered host memory
        static thread_local auto neighbors_ptr = []() {
            void* ptr;
            hipHostMalloc(&ptr, sizeof(int32_t) * 256);
            return reinterpret_cast<int32_t*>(ptr);
        }();

        // Distances are not returned, so we keep them in plain device memory
        static thread_local auto distances_ptr = []() {
            void* ptr;
            hipMalloc(&ptr, sizeof(float) * 256);
            return reinterpret_cast<float*>(ptr);
        }();

        raft::device_resources const& res = raft::device_resources_manager::get_device_resources();

        using namespace cuvs::neighbors;

        // Prepare output arrays
        auto neighbors = raft::make_device_matrix_view<uint32_t, int64_t>(
            reinterpret_cast<uint32_t*>(neighbors_ptr), 1, topk);
        auto distances = raft::make_device_matrix_view<float, int64_t>(distances_ptr, 1, topk);

        // Create an mdspan from the raw pointer
        auto span = raft::make_device_matrix_view<float, int64_t>(query, 1, index->index.dim());

        // Perform the search
        cagra::search_params search_params;
        search_params.itopk_size = raft::bound_by_power_of_two(topk);
        search_params.persistent = true;
        search_params.persistent_device_usage = 0.98;
        search_params.algo = cagra::search_algo::SINGLE_CTA;
        search_params.max_queries = 1000;
        search_params.search_width = 32;
        search_params.max_iterations = 4;

        cagra::search(res, search_params, index->index, raft::make_const_mdspan(span), neighbors, distances);

        return neighbors_ptr;
    }

    void free_cagra_index(jv_cagra_index_t* index) {
        if (index == nullptr) {
            return;
        }
        delete index;
    }

    void save_cagra_index(jv_cagra_index_t* index, const char* filename) {
        if (index == nullptr || filename == nullptr) {
            return;
        }

        raft::device_resources const& res = raft::device_resources_manager::get_device_resources();

        try {
            cuvs::neighbors::cagra::serialize(res, filename, index->index);
        } catch (const std::exception& e) {
            // Handle or log the error
            std::cerr << "Error saving CAGRA index: " << e.what() << std::endl;
        }
    }

    jv_cagra_index_t* load_cagra_index(const char* filename) {
        if (filename == nullptr) {
            return nullptr;
        }

        raft::device_resources const& res = raft::device_resources_manager::get_device_resources();

        try {
            // Create an index with default metric (L2Expanded)
            auto loaded_index = std::make_unique<cuvs::neighbors::cagra::index<float, uint32_t>>(res);

            // Deserialize into the created index
            cuvs::neighbors::cagra::deserialize(res, filename, loaded_index.get());

            // Create and return a new jv_cagra_index_t with the loaded index
            return new jv_cagra_index_t(std::move(*loaded_index));
        } catch (const std::exception& e) {
            // Handle or log the error
            std::cerr << "Error loading CAGRA index: " << e.what() << std::endl;
            return nullptr;
        }
    }
    }
