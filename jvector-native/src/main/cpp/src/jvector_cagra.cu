#include "hip/hip_runtime.h"
/*
 * All changes to the original code are Copyright DataStax, Inc.
 *
 * Please see the included license file for details.
 */

/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <random>
#include <chrono>
#include <iostream>
#include <vector>
#include <mutex>

#include <cstdint>
#include <cstdlib>

#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_resources.hpp>
#include <raft/core/device_resources_manager.hpp>
#include <raft/random/make_blobs.cuh>

#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <cuvs/neighbors/cagra.hpp>

struct jv_cagra_builder_t {
    raft::device_resources dev_resources;
    std::vector<std::vector<float>> host_vectors;
    int64_t dim;

    jv_cagra_builder_t(int32_t n_nodes, int64_t dim)
        : dev_resources(), dim(dim) {
        host_vectors.reserve(n_nodes);
    }
};

struct jv_cagra_index_t {
    cuvs::neighbors::cagra::index<float, uint32_t> index;

    jv_cagra_index_t(cuvs::neighbors::cagra::index<float, uint32_t>&& index)
        : index(std::move(index)) {}
};

extern "C" {
    jv_cagra_builder_t* create_cagra_builder(int32_t n_nodes, int64_t dim) {
        return new jv_cagra_builder_t(n_nodes, dim);
    }

    void add_node(jv_cagra_builder_t* builder, float* vector) {
        if (builder == nullptr || vector == nullptr) {
            return;
        }
        std::vector<float> vec(builder->dim);
        std::copy(vector, vector + builder->dim, vec.begin());
        builder->host_vectors.push_back(std::move(vec));
    }

    jv_cagra_index_t* build_cagra_index(jv_cagra_builder_t* builder) {
        if (builder == nullptr) {
            return nullptr;
        }

        using namespace cuvs::neighbors;
        int64_t n_nodes = builder->host_vectors.size();
        int64_t dim = builder->dim;

        // Create device matrices
        auto device_vectors = raft::make_device_matrix<float, int64_t>(builder->dev_resources, n_nodes, dim);

        // Copy data to device
        std::vector<float> flattened_vectors;
        for (const auto& vec : builder->host_vectors) {
            flattened_vectors.insert(flattened_vectors.end(), vec.begin(), vec.end());
        }
        raft::copy(device_vectors.data_handle(), flattened_vectors.data(), flattened_vectors.size(), raft::resource::get_cuda_stream(builder->dev_resources));
        builder->dev_resources.sync_stream();

        // Build the index
        // DEMOFIXME: tune index_params instead of using defaults
        cagra::index_params index_params;
        auto index = cagra::build(builder->dev_resources, index_params, raft::make_const_mdspan(device_vectors.view()));
        builder->dev_resources.sync_stream();
        return new jv_cagra_index_t(std::move(index));

        // DEMOFIXME: clean up builder resources
    }

    int32_t* search_cagra_index(jv_cagra_index_t* index, float* query, int32_t topk) {
        if (index == nullptr || query == nullptr) {
            return nullptr;
        }

        raft::device_resources const& res = raft::device_resources_manager::get_device_resources();

        using namespace cuvs::neighbors;

        // Prepare output arrays
        auto neighbors = raft::make_device_matrix<uint32_t>(res, 1, topk);
        auto distances = raft::make_device_matrix<float>(res, 1, topk);

        // Create an mdspan from the raw pointer
        auto span = raft::make_device_matrix_view<float, int64_t>(query, 1, index->index.dim());

        // Perform the search
        cagra::search_params search_params;
        search_params.itopk_size = topk;
        cagra::search(res, search_params, index->index, raft::make_const_mdspan(span), neighbors.view(), distances.view());

        // Copy results back to host
        std::vector<uint32_t> host_neighbors(topk);
        raft::copy(host_neighbors.data(), neighbors.data_handle(), topk, res.get_stream());
        raft::resource::sync_stream(res, res.get_stream());

        // DEMOFIXME: eliminate a copy
        // Allocate result array and copy the data
        int32_t* result = static_cast<int32_t*>(malloc(topk * sizeof(int32_t)));
        std::copy(host_neighbors.begin(), host_neighbors.end(), result);

        return result;
    }

    void free_cagra_index(jv_cagra_index_t* index) {
        if (index == nullptr) {
            return;
        }
        delete index;
    }

    void save_cagra_index(jv_cagra_index_t* index, const char* filename) {
        if (index == nullptr || filename == nullptr) {
            return;
        }

        raft::device_resources const& res = raft::device_resources_manager::get_device_resources();

        try {
            cuvs::neighbors::cagra::serialize(res, filename, index->index);
        } catch (const std::exception& e) {
            // Handle or log the error
            std::cerr << "Error saving CAGRA index: " << e.what() << std::endl;
        }
    }

    jv_cagra_index_t* load_cagra_index(const char* filename) {
        if (filename == nullptr) {
            return nullptr;
        }

        raft::device_resources const& res = raft::device_resources_manager::get_device_resources();

        try {
            // Create an index with default metric (L2Expanded)
            auto loaded_index = std::make_unique<cuvs::neighbors::cagra::index<float, uint32_t>>(res);

            // Deserialize into the created index
            cuvs::neighbors::cagra::deserialize(res, filename, loaded_index.get());

            // Create and return a new jv_cagra_index_t with the loaded index
            return new jv_cagra_index_t(std::move(*loaded_index));
        } catch (const std::exception& e) {
            // Handle or log the error
            std::cerr << "Error loading CAGRA index: " << e.what() << std::endl;
            return nullptr;
        }
    }
    }
